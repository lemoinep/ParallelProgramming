#include "hip/hip_runtime.h"
//
// nvcc -O3 -DUSE_DP 03_array_reduce_gpu_atomic.cu
// nvcc -O3          03_array_reduce_gpu_atomic.cu
//
#include "error_checker.cuh"
#include <stdio.h>

#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

const int NUM_REPEATS = 20;
const int NX = 100000000;
const int size_array = sizeof(real) * NX;
const int BLOCK_SIZE = 128;

void timing(const real *d_x);

int main(int argc, const char * argv[])
{

    printf("\n--Beginning of the main function.\n");
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("\nUsing Device %d: %s\n", dev, deviceProp.name);

    real *h_x = (real *) malloc(size_array);
    for (int n = 0; n < NX; ++n)
        h_x[n] = 1.23;
    real *d_x;
    CHECK(hipMalloc(&d_x, size_array));
    CHECK(hipMemcpy(d_x, h_x, size_array, hipMemcpyHostToDevice));

    printf("\n\tUsing atomicAdd:\n");
    timing(d_x);

    free(h_x);
    CHECK(hipFree(d_x));

    printf("\n--Ending of the main function.\n\n");
    return 0;
}


void __global__ reduce(const real *d_x, real *d_y, const int N)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int n = bid * blockDim.x + tid;
    extern __shared__ real s_y[];
    s_y[tid] = (n < NX) ? d_x[n] : 0.0;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (tid < offset)
            s_y[tid] += s_y[tid + offset];
        __syncthreads();
    }

    if (tid == 0)
        atomicAdd(d_y, s_y[0]);
}

real reduce(const real *d_x)
{
    const int grid_size = (NX + BLOCK_SIZE - 1) / BLOCK_SIZE;
    const int smem = sizeof(real) * BLOCK_SIZE;

    real h_y[1] = {0};
    real *d_y;
    CHECK(hipMalloc(&d_y, sizeof(real)));
    CHECK(hipMemcpy(d_y, h_y, sizeof(real), hipMemcpyHostToDevice));

    reduce<<<grid_size, BLOCK_SIZE, smem>>>(d_x, d_y, NX);

    CHECK(hipMemcpy(h_y, d_y, sizeof(real), hipMemcpyDeviceToHost));
    CHECK(hipFree(d_y));

    return h_y[0];
}

void timing(const real *d_x)
{
    real sum = 0;

    for (int repeat = 0; repeat < NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        sum = reduce(d_x); 

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("\tTime = %g ms.\n", elapsed_time);

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    printf("\tSum = %f.\n", sum);
}


